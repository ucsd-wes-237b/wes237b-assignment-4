#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "timer.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
	// TODO: complete the block matrix kernel function
}


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}


float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	Timer gpu_timer;
	gpu_timer.start();

	//TODO: launch the kernel function
	
	CudaCheckError();
	CudaSafeCall(hipDeviceSynchronize());
	gpu_timer.stop();
	float gpu_time = gpu_timer.getElapsed();
	gpu_timer.end();

	return gpu_time;
}


